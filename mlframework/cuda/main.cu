/*
Google Colab
!nvcc -c main.cu -o main
!./main
*/

#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <iostream>
#include <vector>
#include <sys/time.h>
#include <hipblas.h>
#include <hip/library_types.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>

void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};

#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

void randomize_matrix(float *mat, int N) {
  // NOTICE: Use gettimeofday instead of srand((unsigned)time(NULL)); the time
  // precision is too low and the same random number is generated.
  struct timeval time {};
  gettimeofday(&time, nullptr);
  srand(time.tv_usec);
  for (int i = 0; i < N; i++) {
    float tmp = (float)(rand() % 5) + 0.01 * (rand() % 5);
    tmp = (rand() % 2 == 0) ? tmp : tmp * (-1.);
    mat[i] = tmp;
  }
}

bool verify_matrix(float *matRef, float *matOut, int N) {
  double diff = 0.0;
  int i;
  for (i = 0; i < N; i++) {
    diff = std::fabs(matRef[i] - matOut[i]);
    if (diff > 0.01) {
      printf("Divergence! Should %5.2f, Is %5.2f (Diff %5.2f) at %d\n",
             matRef[i], matOut[i], diff, i);
      return false;
    }
  }
  return true;
}

void runCublasFP32(hipblasHandle_t handle, int M, int N, int K, float alpha,
                   float *A, float *B, float beta, float *C) {
  // cuBLAS uses column-major order. So we change the order of our row-major A &
  // B, since (B^T*A^T)^T = (A*B)
  // This runs cuBLAS in full fp32 mode
  hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_32F,
               N, A, HIP_R_32F, K, &beta, C, HIP_R_32F, N, HIPBLAS_COMPUTE_32F,
               CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

void run_kernel(int kernel_num, int M, int N, int K, float alpha, float *A,
                float *B, float beta, float *C, hipblasHandle_t handle) {
  switch (kernel_num) {
  case 0:
    runCublasFP32(handle, M, N, K, alpha, A, B, beta, C);
    break;
  // case 1:
  //   run_sgemm_naive(M, N, K, alpha, A, B, beta, C);
    // break;
  // case 2:
  //   run_sgemm_coalesce(M, N, K, alpha, A, B, beta, C);
  //   break;
  // case 3:
  //   run_sgemm_shared_mem_block(M, N, K, alpha, A, B, beta, C);
  //   break;
  // case 4:
  //   runSgemm1DBlocktiling(M, N, K, alpha, A, B, beta, C);
  //   break;
  // case 5:
  //   runSgemm2DBlocktiling(M, N, K, alpha, A, B, beta, C);
  //   break;
  // case 6:
  //   runSgemmVectorize(M, N, K, alpha, A, B, beta, C);
  //   break;
  // case 7:
  //   runSgemmResolveBankConflicts(M, N, K, alpha, A, B, beta, C);
  //   break;
  // case 8:
  //   runSgemmResolveBankExtraCol(M, N, K, alpha, A, B, beta, C);
  //   break;
  // case 9:
  //   runSgemmAutotuned(M, N, K, alpha, A, B, beta, C);
  //   break;
  // case 10:
  //   runSgemmWarptiling(M, N, K, alpha, A, B, beta, C);
  //   break;
  // case 11:
  //   runSgemmDoubleBuffering(M, N, K, alpha, A, B, beta, C);
  //   break;
  // case 12:
  //   runSgemmDoubleBuffering2(M, N, K, alpha, A, B, beta, C);
  //   break;
  default:
    throw std::invalid_argument("Unknown kernel number");
  }
}

int main(int argc, char **argv) {
    hipblasHandle_t handle;
    if (hipblasCreate(&handle)) {
      std::cerr << "Create cublas handle error." << std::endl;
      exit(EXIT_FAILURE);
    };

    float elapsed_time;
    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);
    
    long m, n, k, max_size;
    max_size = 4096;

    float alpha = 0.5, beta = 3.0; // GEMM input parameters, C=α*AB+β*C

    float *A = nullptr, *B = nullptr, *C = nullptr,
      *C_ref = nullptr; // host matrices
    float *dA = nullptr, *dB = nullptr, *dC = nullptr,
      *dC_ref = nullptr; // device matrices

    A = (float *)malloc(sizeof(float) * max_size * max_size);
    B = (float *)malloc(sizeof(float) * max_size * max_size);
    C = (float *)malloc(sizeof(float) * max_size * max_size);
    C_ref = (float *)malloc(sizeof(float) * max_size * max_size);

    randomize_matrix(A, max_size * max_size);
    randomize_matrix(B, max_size * max_size);
    randomize_matrix(C, max_size * max_size);

    cudaCheck(hipMalloc((void **)&dA, sizeof(float) * max_size * max_size));
    cudaCheck(hipMalloc((void **)&dB, sizeof(float) * max_size * max_size));
    cudaCheck(hipMalloc((void **)&dC, sizeof(float) * max_size * max_size));
    cudaCheck(hipMalloc((void **)&dC_ref, sizeof(float) * max_size * max_size));
  
    cudaCheck(hipMemcpy(dA, A, sizeof(float) * max_size * max_size,
                        hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dB, B, sizeof(float) * max_size * max_size,
                        hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dC, C, sizeof(float) * max_size * max_size,
                        hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dC_ref, C, sizeof(float) * max_size * max_size,
                       hipMemcpyHostToDevice));

    int kernel_num = 0;

    int size = 1024;
    m = n = k = size;
    run_kernel(0, m, n, k, alpha, dA, dB, beta, dC_ref, handle); // cuBLAS
    run_kernel(kernel_num, m, n, k, alpha, dA, dB, beta, dC, handle); // Executes the kernel, modifies the result matrix
    if (!verify_matrix(C_ref, C, m * n)) {
      std::cout << "Failed to pass the correctness verification against NVIDIA cuBLAS." << std::endl;
    }

    hipEventRecord(beg);
    // run kernel here
    run_kernel(kernel_num, m, n, k, alpha, dA, dB, beta, dC, handle);
    hipEventRecord(end);
    
    hipEventSynchronize(beg);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, beg, end);
    elapsed_time /= 1000.; // Convert to seconds
    printf("Elapsed time: (%7.9f) s\n", elapsed_time);
}