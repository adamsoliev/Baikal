
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <iostream>
#include <vector>


int main(int argc, char **argv) {
    float elapsed_time;
    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);

    hipEventRecord(beg);
    // run kernel here
    hipEventRecord(end);
    
    hipEventSynchronize(beg);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, beg, end);
    elapsed_time /= 1000.; // Convert to seconds
    printf("Elapsed time: (%7.9f) s\n", elapsed_time);
}